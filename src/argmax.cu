#include "utils.cuh"
#include <cmath>
#include <cstdio>
#include <hipcub/hipcub.hpp>

struct MaskingOp {
    const float* data;
    const bool* mask;

    __device__ float operator()(const int& i) const {
        return mask[i] ? data[i] : -INFINITY;
    }
};

extern "C" int argmax(float* data, bool* mask, size_t size) {
    using namespace hipcub;

    // 1. Create a counting iterator [0, 1, 2, ..., size-1]
    CountingInputIterator<int> counting_iter(0);

    // 2. Create the transform iterator that applies masking on-the-fly
    MaskingOp op = {data, mask};
    TransformInputIterator<float, MaskingOp, CountingInputIterator<int>>
        masked_iter(counting_iter, op);

    // 3. Allocate output
    KeyValuePair<int, float>* d_output;
    hipMalloc(&d_output, sizeof(KeyValuePair<int, float>));

    // 4. Temp storage
    void* d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;

    // First call to get temp storage size
    DeviceReduce::ArgMax(d_temp_storage, temp_storage_bytes, masked_iter, d_output, size);
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    // 5. Actual ArgMax call
    DeviceReduce::ArgMax(d_temp_storage, temp_storage_bytes, masked_iter, d_output, size);

    // 6. Copy result back
    KeyValuePair<int, float> h_output;
    hipMemcpy(&h_output, d_output, sizeof(KeyValuePair<int, float>), hipMemcpyDeviceToHost);

    // 7. Free
    hipFree(d_output);
    hipFree(d_temp_storage);

    return h_output.key;
}
